#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/driver_types.h>
#include <hiprand.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <cstdio>
#include <hip/hip_runtime.h>

#include "support.h"
#include "hash_kernel.cu"
#include "nonce_kernel.cu"
#include "reduction_kernel.cu"

// to activate debug statements
#define DEBUG 1

// program constants
#define BLOCK_SIZE 1024
#define SEED 123

// solution constants
#define MAX 123123123
#define TARGET 20

// functions used
unsigned int generate_hash(unsigned int nonce, unsigned int index, unsigned int *transactions, unsigned int n_transactions);
void read_file(char *file, unsigned int *transactions, unsigned int n_transactions);
void err_check(hipError_t ret, char *msg, int exit_code);

/* Main ------------------ //
 *   This is the main program.
 */
int main(int argc, char *argv[])
{
    // Catch console errors
    if (argc != 6)
    {
        printf("USE LIKE THIS: gpu_mining transactions.csv n_transactions trials out.csv time.csv\n");
        return EXIT_FAILURE;
    }

    // Output files
    FILE *output_file = fopen(argv[4], "w");
    FILE *time_file = fopen(argv[5], "w");

    // Read in the transactions
    unsigned int n_transactions = strtoul(argv[2], NULL, 10);
    unsigned int *transactions = (unsigned int *)calloc(n_transactions, sizeof(unsigned int));
    read_file(argv[1], transactions, n_transactions);

    // get the number of trials
    unsigned int trials = strtoul(argv[3], NULL, 10);

    // -------- Start Mining ------------------------------------------------------- //
    // ----------------------------------------------------------------------------- //

    // Set timer and cuda error return
    Timer timer;
    startTime(&timer);
    hipError_t cuda_ret;

    // To use with kernels
    int num_blocks = ceil((float)trials / (float)BLOCK_SIZE);
    dim3 dimGrid(num_blocks, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);

    // ------ Step 1: generate the nonce values ------ //

    // Allocate the nonce device memory
    unsigned int *device_nonce_array;
    cuda_ret = hipMalloc((void **)&device_nonce_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char *)"Unable to allocate nonces to device memory!", 1);

    // Launch the nonce kernel
    nonce_kernel<<<dimGrid, dimBlock>>>(
        device_nonce_array, // put nonces into here
        trials,             // size of array
        MAX,                // to mod with
        SEED                // random seed
    );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char *)"Unable to launch nonce kernel!", 2);

    // Get nonces from device memory
    unsigned int *nonce_array = (unsigned int *)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(nonce_array, device_nonce_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char *)"Unable to read nonce from device memory!", 3);

    // ------ Step 2: Generate the hash values ------ //
    // Allocate memory for transactions on the device
    unsigned int *device_transactions;
    cuda_ret = hipMalloc((void **)&device_transactions, n_transactions * sizeof(unsigned int));
    err_check(cuda_ret, (char *)"Unable to allocate transactions to device memory!", 4);
    cuda_ret = hipMemcpy(device_transactions, transactions, n_transactions * sizeof(unsigned int), hipMemcpyHostToDevice);
    err_check(cuda_ret, (char *)"Unable to copy transactions to device memory!", 5);

    // Allocate memory for hash array on the device
    unsigned int *device_hash_array;
    cuda_ret = hipMalloc((void **)&device_hash_array, trials * sizeof(unsigned int));
    err_check(cuda_ret, (char *)"Unable to allocate hash array to device memory!", 6);

    // Launch the hash kernel
    hash_kernel<<<dimGrid, dimBlock>>>(
        device_hash_array,   // put hash values into here
        device_nonce_array,  // nonce array
        trials,              // size of array
        device_transactions, // transactions array
        n_transactions,      // number of transactions
        MAX                  // modulus value for hash function
    );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char *)"Unable to launch hash kernel!", 7);

    // Copy the hash array from device to host
    unsigned int *hash_array = (unsigned int *)calloc(trials, sizeof(unsigned int));
    cuda_ret = hipMemcpy(hash_array, device_hash_array, trials * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char *)"Unable to read hash array from device memory!", 8);

    // Free memory
    free(transactions);
    hipFree(device_transactions);
    hipFree(device_nonce_array);
    hipFree(device_hash_array);

    // ------ Step 3: Find the nonce with the minimum hash value ------ //
    // Allocate memory for local min hash and nonce arrays on the device
    unsigned int *device_min_hash_array;
    unsigned int *device_min_nonce_array;
    cuda_ret = hipMalloc((void **)&device_min_hash_array, num_blocks * sizeof(unsigned int));
    err_check(cuda_ret, (char *)"Unable to allocate min hash array to device memory!", 9);
    cuda_ret = hipMalloc((void **)&device_min_nonce_array, num_blocks * sizeof(unsigned int));
    err_check(cuda_ret, (char *)"Unable to allocate min nonce array to device memory!", 10);

    // Launch the reduction kernel
    reduction_kernel<<<dimGrid, dimBlock, 2 * BLOCK_SIZE * sizeof(unsigned int)>>>(
        device_hash_array,
        device_nonce_array,
        device_min_hash_array,
        device_min_nonce_array,
        trials
    );
    cuda_ret = hipDeviceSynchronize();
    err_check(cuda_ret, (char *)"Unable to launch reduction kernel!", 11);

    // Copy local min hash and nonce arrays from device to host
    unsigned int *min_hash_array = (unsigned int *)calloc(num_blocks, sizeof(unsigned int));
    unsigned int *min_nonce_array = (unsigned int *)calloc(num_blocks, sizeof(unsigned int));
    cuda_ret = hipMemcpy(min_hash_array, device_min_hash_array, num_blocks * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char *)"Unable to read min hash array from device memory!", 12);
    cuda_ret = hipMemcpy(min_nonce_array, device_min_nonce_array, num_blocks * sizeof(unsigned int), hipMemcpyDeviceToHost);
    err_check(cuda_ret, (char *)"Unable to read min nonce array from device memory!", 13);

    // Find the global minimum hash and corresponding nonce
    unsigned int min_hash = UINT_MAX;
    unsigned int min_nonce = 0;
    for (int i = 0; i < num_blocks; i++) {
        if (min_hash_array[i] < min_hash) {
            min_hash = min_hash_array[i];
            min_nonce = min_nonce_array[i];
        }
    }

    // Free memory
    free(nonce_array);
    free(hash_array);

    stopTime(&timer);
    // ----------------------------------------------------------------------------- //
    // -------- Finish Mining ------------------------------------------------------ //

    // Get if suceeded
    char *res = (char *)malloc(8 * sizeof(char));
    if (min_hash < TARGET)
        res = (char *)"Success!";
    else
        res = (char *)"Failure.";

    // Show results in console
    if (DEBUG)
        printf("%s\n   Min hash:  %u\n   Min nonce: %u\n   %f seconds\n",
               res,
               min_hash,
               min_nonce,
               elapsedTime(timer));

    // Print results
    fprintf(output_file, "%s\n%u\n%u\n", res, min_hash, min_nonce);
    fprintf(time_file, "%f\n", elapsedTime(timer));

    // Cleanup
    fclose(time_file);
    fclose(output_file);

    return 0;
} // End Main -------------------------------------------- //

/* Read File -------------------- //
 *   Reads in a file of transactions.
 */
void read_file(char *file, unsigned int *transactions, unsigned int n_transactions)
{

    // open file
    FILE *trans_file = fopen(file, "r");
    if (trans_file == NULL)
        fprintf(stderr, "ERROR: could not read the transaction file.\n"),
            exit(-1);

    // read items
    char line[100] = {0};
    for (int i = 0; i < n_transactions && fgets(line, 100, trans_file); ++i)
    {
        char *p;
        transactions[i] = strtof(line, &p);
    }

    fclose(trans_file);

} // End Read File ------------- //

/* Error Check ----------------- //
 *   Exits if there is a CUDA error.
 */
void err_check(hipError_t ret, char *msg, int exit_code)
{
    if (ret != hipSuccess)
        fprintf(stderr, "%s \"%s\".\n", msg, hipGetErrorString(ret)),
            exit(exit_code);
} // End Error Check ----------- //
